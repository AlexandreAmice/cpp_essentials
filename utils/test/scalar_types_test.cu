#include "hip/hip_runtime.h"
#include <string>
#include <type_traits>

#include "utils/scalar_types.h"
#include <gtest/gtest.h>

namespace common_utils {
namespace {
__global__ void checkCommonUtilsScalarTypes(bool* results) {
  results[0] = CommonUtilsScalarType<float>;
  results[1] = CommonUtilsScalarType<double>;
}
// Test that concept works for supported types in CUDA
TEST(ScalarTypesTest, ConceptSupportsFloatAndDoubleInCuda) {
  bool h_results[2];
  bool* d_results;
  hipMalloc(&d_results, 2 * sizeof(bool));

  // Call the global kernel function instead of using a lambda directly
  checkCommonUtilsScalarTypes<<<1, 1>>>(d_results);
  hipDeviceSynchronize();

  hipMemcpy(h_results, d_results, 2 * sizeof(bool), hipMemcpyDeviceToHost);
  hipFree(d_results);

  EXPECT_TRUE(h_results[0]);
  EXPECT_TRUE(h_results[1]);
}

template <CommonUtilsScalarType T>
__global__ void deviceTestKernel(bool* result, T input);

template <typename T>
struct CudaTemplateTest {
  // Device function that can be called from kernels
  __device__ static bool DeviceTest(T value) { return value > T(0); }

  // Definition of our device kernel function

  // Host function that launches a kernel to test the device function
  static bool RunTest(T value) {
    bool h_result;
    bool* d_result;
    hipMalloc(&d_result, sizeof(bool));

    // Launch the kernel
    deviceTestKernel<T><<<1, 1>>>(d_result, value);
    hipDeviceSynchronize();

    // Get the result
    hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_result);

    return h_result;
  }
};
template <CommonUtilsScalarType T>
__global__ void deviceTestKernel(bool* result, T input) {
  *result = CudaTemplateTest<T>::DeviceTest(input);
}

// Use the macro to instantiate the template for all supported types
INSTANTIATE_FOR_SUPPORTED_SCALARS(CudaTemplateTest);

// Test that the template was instantiated correctly for all types
TEST(ScalarTypesTest, CudaInstantiationWorks) {
  EXPECT_TRUE(CudaTemplateTest<float>::RunTest(1.0f));
  EXPECT_FALSE(CudaTemplateTest<float>::RunTest(-1.0f));
  EXPECT_TRUE(CudaTemplateTest<double>::RunTest(2.0));
  EXPECT_FALSE(CudaTemplateTest<double>::RunTest(-2.0));
}

// CUDA device function template
template <typename T>
__device__ T cuda_multiply_by_two(T value) {
  return value * T(2);
}

template <typename T>
__global__ void cuda_multiply_by_two_kernel(T* result, T input) {
  *result = cuda_multiply_by_two(input);
}

// Host wrapper function template that launches a kernel
template <typename T>
T cuda_wrapper(T value) {
  T h_result;
  T* d_result;
  hipMalloc(&d_result, sizeof(T));

  // Launch the kernel
  cuda_multiply_by_two_kernel<<<1, 1>>>(d_result, value);
  hipDeviceSynchronize();

  // Get the result
  hipMemcpy(&h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
  hipFree(d_result);

  return h_result;
}

// Use the macro to instantiate the function for all supported types
INSTANTIATE_FUNCTION_FOR_SCALAR_TYPES(cuda_wrapper);

// Test that the function was instantiated correctly for all types
TEST(ScalarTypesTest, CudaFunctionInstantiationWorks) {
  EXPECT_FLOAT_EQ(cuda_wrapper(2.5f), 5.0f);  // Test with float
  EXPECT_DOUBLE_EQ(cuda_wrapper(3.0), 6.0);   // Test with double
}
}  // namespace
}  // namespace common_utils
